#include "hip/hip_runtime.h"
#include "caffe/util/unsupervised_learner.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
bool PCALearner<Dtype>::step_gpu(const vector<shared_ptr<Blob<Dtype> > >&  input, Dtype* objective) {
  CHECK_GE(input.size(), 1);
  CHECK(input[0]) << "step_gpu input was null";
  CHECK_GT(input[0]->count(), 0) << "step_gpu input has data";
  if (objective) {
    *objective = INFINITY;
  }
  if (!called_setup_) {
    this->setup(input);
    called_setup_ = true;
  }
  if (!calculated_mean_) {
    update_mean_gpu(input[0]);
    iter_++;
    calculated_mean_ = iter_ >= num_batches_;
    return true;
  }
  if (iter_ < num_batches_ * 2) {
    update_covariance_gpu(input[0]);
    iter_++;
  }
  if (iter_ == num_batches_ * 2) {
    calc_pca_gpu();
  }
  return iter_ < num_batches_ * 2;
}

template <typename Dtype>
void PCALearner<Dtype>::fill_gpu(const vector<shared_ptr<Blob<Dtype> > >& blobs) {
  CHECK_EQ(blobs.size(), 2);
  CHECK_EQ(blobs[0]->count(), out_dim_ * dim_);
  CHECK_EQ(blobs[1]->count(), out_dim_);
  caffe_copy(out_dim_ * dim_, P_.gpu_data(), blobs[0]->mutable_gpu_data());
  caffe_gpu_gemv<Dtype>(CblasNoTrans, out_dim_, dim_,
    Dtype(-1.0), P_.gpu_data(), mean_.gpu_data(),
    Dtype(0.0), blobs[1]->mutable_gpu_data());
}

template <typename Dtype>
Dtype PCALearner<Dtype>::objective_gpu(const vector<shared_ptr<Blob<Dtype> > >& input) {
  return INFINITY;
}

template <typename Dtype>
void PCALearner<Dtype>::update_mean_gpu(const shared_ptr<Blob<Dtype> >& input) {
  const Dtype* data = input->gpu_data();
  caffe_gpu_gemv<Dtype>(CblasTrans, batch_size_, dim_,
    Dtype(1.0 / ((iter_ + 1) * batch_size_)), data,
    sum_multiplier_.gpu_data(), Dtype(iter_ / (iter_ + 1.0)),
    mean_.mutable_gpu_data());
}

template <typename Dtype>
void PCALearner<Dtype>::update_covariance_gpu(const shared_ptr<Blob<Dtype> >& input) {
  Dtype* data = input->mutable_gpu_data();
  // Subtract the mean from data
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, batch_size_, dim_, 1,
    (Dtype)(-1.0), sum_multiplier_.gpu_data(), mean_.gpu_data(),
    (Dtype)1., data);
  // Construct partial covariance matrix
  const int cov_iter = iter_ - num_batches_;
  caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, dim_, dim_, batch_size_,
    (Dtype)(1.0 / ((cov_iter + 1) * batch_size_ - 1.0)), data, data,
    (Dtype)((cov_iter * batch_size_ - 1.0) / ((cov_iter + 1) * batch_size_ - 1.0)), cov_.mutable_gpu_data());
}

template <typename Dtype>
void PCALearner<Dtype>::calc_pca_gpu() {
  // Comment containing debug code...
  // const Dtype* mean_cpu = mean_.cpu_data();
  // ostringstream ss2;
  // for (int i = 0; i < 10; ++i) {
  //   ss2 << mean_cpu[i] << ", ";
  // }
  // LOG(INFO) << "Mean: " << ss2.str();
  // const Dtype* Xcpu = cov_.cpu_data();
  // for (int i = 0; i < 3; ++i) {
  //   ostringstream ss;
  //   for (int j = 0; j < 3; ++j) {
  //     ss << Xcpu[i * dim_ + j] << ", ";
  //   }
  //   LOG(INFO) << "Cov: " << ss.str();
  // }

  const Dtype* X = cov_.gpu_data();
  Dtype* R = cov_.mutable_gpu_diff();
  Dtype* U = P_.mutable_gpu_data();
  Dtype* V = P_.mutable_gpu_diff();
  Blob<Dtype> lambda, temp;
  lambda.Reshape(out_dim_, 1, 1, 1);
  Dtype* L = lambda.mutable_cpu_data();
  temp.Reshape(out_dim_, 1, 1, 1);
  Dtype* A = temp.mutable_gpu_data();
  Dtype* B = temp.mutable_gpu_diff();

  // Using the GS-PCA algorithm as presented in
  // Parallel GPU Implementation of Iterative PCA Algorithms (2008)
  // by M. Andrecut

  // input: X, MxN matrix (data)
  // input: M = number of rows in X
  // input: N = number of columns in X
  // input: K = number of components (K<=N)
  // output: T, MxK scores matrix // output: P, NxK loads matrix // output: R, MxN residual matrix
  // Note: remeber that BLAS expects column-major matrices vs. Caffe's row-major matrices.

  const int J = 10000; // max number of powers... TODO: switch to parameter
  const int min_J = 0; // min number of powers...
  CHECK_LE(min_J, J);
  Dtype er = 1.0e-7; // max error
  caffe_copy(dim_ * dim_, X, R);
  for (int k = 0; k < out_dim_; ++k) {
    Dtype mu = 0.0;
    caffe_copy(dim_, R + k * dim_, V + k * dim_);
    int j;
    for (j = 0; j < J; ++j) {
      caffe_gpu_gemv<Dtype>(CblasNoTrans, dim_, dim_,
        Dtype(1.0), R, V + k * dim_,
        Dtype(0.0), U + k * dim_);
      //hipblasDgemv (’t’, dim_, dim_, 1.0, dR, dim_, &dT[k*dim_], 1, 0.0, &dP[k*dim_], 1);
      if (k > 0) {
        caffe_gpu_gemv<Dtype>(CblasNoTrans, k, dim_,
          Dtype(1.0), U, U + k * dim_,
          Dtype(0.0), A);
        caffe_gpu_gemv<Dtype>(CblasTrans, k, dim_,
          Dtype(-1.0), U, A,
          Dtype(1.0), U + k * dim_);
        // hipblasDgemv (’t’, dim_, k, 1.0, dP, dim_, &dP[k*dim_], 1, 0.0, dU, 1);
        // hipblasDgemv (’n’, dim_, k, -1.0, dP, dim_, dU, 1, 1.0, &dP[k*dim_], 1);
      }
      caffe_gpu_scal(dim_, Dtype(1.0 / caffe_gpu_nrm2(dim_, U + k * dim_)), U + k * dim_);
      // hipblasDscal (dim_, 1.0/hipblasDnrm2(dim_, &dP[k*dim_], 1), &dP[k*dim_], 1);
      caffe_gpu_gemv<Dtype>(CblasTrans, dim_, dim_,
        Dtype(1.0), R, U + k * dim_,
        Dtype(0.0), V + k * dim_);
      // hipblasDgemv (’n’, dim_, dim_, 1.0, dR, dim_, &dP[k*dim_], 1, 0.0, &dT[k*dim_], 1);
      if (k > 0) {
        caffe_gpu_gemv<Dtype>(CblasNoTrans, k, dim_,
          Dtype(1.0), V, V + k * dim_,
          Dtype(0.0), B);
        caffe_gpu_gemv<Dtype>(CblasTrans, k, dim_,
          Dtype(-1.0), V, B,
          Dtype(1.0), V + k * dim_);
        // hipblasDgemv (’t’, dim_, k, 1.0, dT, dim_, &dT[k*dim_], 1, 0.0, dU, 1);
        // hipblasDgemv (’n’, dim_, k, -1.0, dT, dim_, dU, 1, 1.0, &dT[k*dim_], 1);
      }
      L[k] = caffe_gpu_nrm2(dim_, V + k * dim_);
      // L[k] = hipblasDnrm2(dim_, &dT[k*dim_], 1);
      caffe_gpu_scal(dim_, Dtype(1.0/L[k]), V + k * dim_);
      // hipblasDscal(dim_, 1.0/L[k], &dT[k*dim_], 1);
      if (fabs(L[k] - mu) < er * L[k] && j >= min_J) {
        break;
      }
      mu = L[k];
    }
    LOG(INFO) << "Lambda #" << k << ": " << L[k] << " (" << j << " iterations)";
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, dim_, dim_, 1,
      -L[k], V + k * dim_, U + k * dim_,
      (Dtype)1., R);
    // hipblasDger (dim_, dim_, - L[k], &dT[k*dim_], 1, &dP[k*dim_], 1, dR, dim_);
  }
  // Calculate P = (D^-0.5) * U^T if whitening is needed
  if (apply_whitening_) {
    if (zca_whitening_) {
      caffe_copy(dim_ * dim_, U, V);
    }
    for (int k = 0; k < out_dim_; k++) {
      caffe_gpu_scal(dim_, Dtype(1.0 / sqrt(L[k] + fudge_factor_)), U + k * dim_);
    }
    if (zca_whitening_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, dim_, dim_, dim_,
        Dtype(1.0), V, U,
        Dtype(0.0), R);
      caffe_copy(dim_ * dim_, R, U);
    }
  }
}

template void PCALearner<float>::fill_gpu(const vector<shared_ptr<Blob<float> > >& blobs);
template void PCALearner<double>::fill_gpu(const vector<shared_ptr<Blob<double> > >& blobs);
template bool PCALearner<float>::step_gpu(const vector<shared_ptr<Blob<float> > >& input, float* objective);
template bool PCALearner<double>::step_gpu(const vector<shared_ptr<Blob<double> > >& input, double* objective);
template float PCALearner<float>::objective_gpu(const vector<shared_ptr<Blob<float> > >& input);
template double PCALearner<double>::objective_gpu(const vector<shared_ptr<Blob<double> > >& input);
}  // namespace caffe
